#include "hip/hip_runtime.h"
// nvcc opengl.cu -lGL -lGLU -lGLEW -lglut

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>
#include <string.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include "../lib/cuPrintf.cu"

using namespace std;

const int32_t GLOBAL_WIDTH = 1024;
const int32_t GLOBAL_HEIGHT = 648;
const uint32_t VAR_COUNT = 2;
const uint32_t FUNC_MAXIMUMS_CNT = 4;
const uint32_t POINTS_COUNT = 2000;
const double SCALE_CHANGE_SPEED = 1.05;
const double PI = 3.1415926;
const double POINT_RADIUS = 10.;
const double DOUBLE_GEN_ACCURACY = 1000.;
const double POINTS_GEN_WIDTH = 10.;
const double POINTS_GEN_HEIGHT = 10.;

const double INERTIA = .08;
const double PARAM_A_GLOBAL = .09;
const double PARAM_A_LOCAL = .03;

const double SHIFT_SPEED_X = 0.;
const double SHIFT_SPEED_Y = 0.;

/*const double FUNC_A[FUNC_MAXIMUMS_CNT][VAR_COUNT] = {
	{2.54, 6.35},
	{7.56, 3.35},
	{7.35, 3.65}
};*/

dim3 blocks2D(32, 32), threads2D(16, 16);
dim3 blocks1D(1024), threads1D(256);

#define CSC(call) {														\
	 hipError_t err = call;												\
	 if(err != hipSuccess) {											\
		  fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
				__FILE__, __LINE__, hipGetErrorString(err));			\
		  exit(1);														\
	 }																	\
} while (0)


const double 	xc = 0.0f,
				yc = 0.0f,
				sx = 5.0f,
				sy = sx * GLOBAL_HEIGHT / GLOBAL_WIDTH,
				FUNC_MIN = 0.,
				FUNC_MAX = .84;


struct Comparator {
	__host__ __device__ bool operator()(double a, double b) {
		return a < b;
	}
};

struct Position {
	double X;
	double Y;
};

__device__ __host__ void setPosition(Position *pos, double x, double y) {
	pos->X = x;
	pos->Y = y;
}

struct Point {
	Position Pos;

	double Angle;
	double Speed;
	double LocalMin;
	Position LocalMinPos;
	uchar4 Pixel;
	//bool PixelGradient;
};

struct GlobalData {
	Point *PointsArr;
	double Min;
	Position MinPos;
	double PointSelectCoeff;
};

GlobalData *GLOBAL;
double *MAX_ARR;

__host__ double fRand(double fMin, double fMax)
{
    double f = (double) (rand() % (int32_t) fMax);
    return fMin + f * (fMax - fMin);
}

__host__ void setGlobalData() {
	CSC(hipMalloc((void**) &GLOBAL, sizeof(GlobalData)));
	CSC(hipMalloc((void**) &MAX_ARR, sizeof(double) * POINTS_COUNT));
	CSC(hipMemset(MAX_ARR, 0., sizeof(double) * POINTS_COUNT));

	Point *tmpPointsArr;
	CSC(hipMalloc((void**) &tmpPointsArr, sizeof(Point) * POINTS_COUNT));
	
	GlobalData globalData;
	globalData.PointsArr = tmpPointsArr;
	globalData.Min = 0;

	CSC(hipMemcpy(GLOBAL, &globalData, sizeof(GlobalData), hipMemcpyHostToDevice));

}

__host__ void destroyGlobalData() {
	GlobalData globalData;
	CSC(hipMemcpy(&globalData, GLOBAL, sizeof(GlobalData), hipMemcpyDeviceToHost));
	Point *tmpPointsArr = globalData.PointsArr;

	CSC(hipFree(tmpPointsArr));
	CSC(hipFree(GLOBAL));
	CSC(hipFree(MAX_ARR));
}

__device__ double distance(double x1, double y1, double x2, double y2) {
	return sqrt(pow(abs(x1 - x2), 2.) + pow(abs(y1 - y2), 2.));
}

__device__ int32_t distance(int32_t x1, int32_t y1, int32_t x2, int32_t y2) {
	return (abs(x1 - x2) + abs(y1 - y2));
}
	
__device__ double func(double x, double y, double t) {
	//return sin(x * x + t) + cos(y * y + t * 0.6) + sin(x * x + y * y + t * 0.3);
	//x /= 10.;
	//y /= 10.;
	double func_a[FUNC_MAXIMUMS_CNT][VAR_COUNT] = {
		{.054, 1.035},
		{3.956, .135},
		{.535, 1.065},
		{1.032, .121}
	};
	double summ = 0.;
	for (uint32_t i = 0; i < FUNC_MAXIMUMS_CNT; i++) {
		summ += 1. / (pow((x - t * SHIFT_SPEED_X) - func_a[i][0], 2.) + pow((y + t * SHIFT_SPEED_Y) - func_a[i][1], 2.));
	}
	//summ = 2.5;
	//cout << summ << endl;
	//cuPrintf("%lf\n", summ);
	return summ;
}

__device__ double getCoordinateX(int32_t i, double scale) {
	return (2.0f * i / (double)(GLOBAL_WIDTH - 1) - 1.0f) * scale;
}

__device__ double getCoordinateY(int32_t j, double scale) {
	return (2.0f * j / (double)(GLOBAL_HEIGHT - 1) - 1.0f) * scale;
}

__device__ int32_t getPixelX(double x, double scale) {
	return (x / (2.0f * scale) + 0.5f) * (double)(GLOBAL_WIDTH - 1);
}

__device__ int32_t getPixelY(double y, double scale) {
	return (y / (2.0f * scale) + 0.5f) * (double)(GLOBAL_HEIGHT - 1);
}

__device__ double func(int32_t i, int32_t j, double t, double scale)  {
	double x = getCoordinateX(i, scale);
	double y = getCoordinateY(j, scale);

	return func(x * sx + xc, -y * sy + yc, t);	 
}

__device__ uchar4 get_color(float f) {
	//f /= 10;
	//f += .1;
	float k = 1.0 / 6.0;
	if (f < k)
		return make_uchar4((int)(f * 255 / k), 0, 0, 0);
	if (f < 2 * k)
		return make_uchar4(255, (int)((f - k) * 255 / k), 0, 0);
	if (f < 3 * k)
		return make_uchar4(255, 255, (int)((f - 2 * k) * 255 / k), 0);
	if (f < 4 * k)
		return make_uchar4(255 - (int)((f - 3 * k) * 255 / k), 255, 255, 0);
	if (f < 5 * k)
		return make_uchar4(0, 255 - (int)((f - 4 * k) * 255 / k), 255, 0);
	if (f < 6 * k)
		return make_uchar4(0, 0, 255 - (int)((f - 5 * k) * 255 / k), 0);
	return make_uchar4(0, 0, 0, 0);
}

__device__ uchar4 getPixel(double t) {
	return make_uchar4((int)(255 * cos(t + 2.)), (int)(255 * cos(t)), (int)(255 * sin(t)), 0);
}

/*__device__ uchar4 getPixel(GlobalData *Global, int32_t i, int32_t j, double f, double t, double scale) {
	for (int32_t n = 0; n < POINTS_COUNT; n++) {
		if (distance(getCoordinateX(i, scale), getCoordinateY(j, scale),
				Global->PointsArr[n].Pos.X, Global->PointsArr[n].Pos.Y) < POINT_RADIUS) {
			return getPixel(t);
		}
		if (distance(getCoordinateX(i, scale), getCoordinateY(j, scale),
				Global->PointsArr[n].Pos.X, Global->PointsArr[n].Pos.Y) < 2. * POINT_RADIUS) {
			return make_uchar4(0, 0, 0, 0);
		}
	}
	cuPrintf("UNIT %d :: %d\n", i, getPixelX(getCoordinateX(i, scale), scale));
	return get_color(f);
}*/


__global__ void drawMap(GlobalData *Global, uchar4* data, double t, double scale) {
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t idy = blockIdx.y * blockDim.y + threadIdx.y;
	int32_t offsetx = blockDim.x * gridDim.x;
	int32_t offsety = blockDim.y * gridDim.y;
	int32_t i, j;

	for(i = idx; i < GLOBAL_WIDTH; i += offsetx) {
		for(j = idy; j < GLOBAL_HEIGHT; j += offsety) {
			double f = (func(i, j, t, scale) - FUNC_MIN) / (FUNC_MAX - FUNC_MIN);
			//data[j * GLOBAL_WIDTH + i] = getPixel(Global, i, j, f, t, scale);
			data[j * GLOBAL_WIDTH + i] = get_color(f); //make_uchar4(0, 0, (int)(f * 255), 255);
		}
	}
	__syncthreads();
}

__global__ void drawPoints(GlobalData *Global, uchar4 *data, double t, double scale) {
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t offsetx = blockDim.x * gridDim.x;
	int32_t i, j;

	for (int32_t n = idx; n < POINTS_COUNT; n += offsetx) {
		i = getPixelX(Global->PointsArr[n].Pos.X, scale);
		j = getPixelY(Global->PointsArr[n].Pos.Y, scale);

		if (i > 0 && j > 0 && i < GLOBAL_WIDTH && j < GLOBAL_HEIGHT) {
			data[j * GLOBAL_WIDTH + i] = getPixel(t);
		}
		//cuPrintf("Unit: %lf :: %lf :: %d\n", getCoordinateX(i, scale), Global->PointsArr[n].Pos.X, i);
	}
}

__device__ void generatePoint(Point *point, int32_t *rand_arr, int32_t num) {
	//hiprandCreateGenerator(&rand_gen ,HIPRAND_RNG_PSEUDO_DEFAULT);

	//point->Pos.X = cuRand() % GLOBAL_WIDTH;
	//point->Pos.Y = cuRand() % GLOBAL_HEIGHT;
	/*hiprandGenerateUniformDouble(rand_gen, &(point->Pos.X), 1);
	hiprandGenerateUniformDouble(rand_gen, &(point->Pos.Y), 1);
	hiprandGenerateUniformDouble(rand_gen, &(point->Angle), 1);*/
	

	point->Pos.X = (double)(rand_arr[3 * num]) / DOUBLE_GEN_ACCURACY * POINTS_GEN_WIDTH;
	point->Pos.Y = (double)(rand_arr[3 * num + 1]) / DOUBLE_GEN_ACCURACY * POINTS_GEN_HEIGHT;
	point->Angle = (double)(rand_arr[3 * num + 2]) / DOUBLE_GEN_ACCURACY * 2. * PI;

	if (rand_arr[3 * num] % 2) {
		point->Pos.X = -point->Pos.X;
	}
	if (rand_arr[3 * num + 1] % 2) {
		point->Pos.Y = -point->Pos.Y;
	}
	point->Speed = 1.;
	point->LocalMin = 0.;
	point->LocalMinPos.X = 0.;
	point->LocalMinPos.Y = 0.;
	point->Pixel = make_uchar4(0, 255, 0, 0);


	//hiprandDestroyGenerator(rand_gen);
}

__global__ void setGlobalDataValues(GlobalData *Global, int32_t *rand_arr) {
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t offsetx = blockDim.x * gridDim.x;

	Global->PointSelectCoeff = rand_arr[0] / DOUBLE_GEN_ACCURACY;
	Global->MinPos.X = 0.;
	Global->MinPos.Y = 0.;

	for (int32_t i = idx; i < POINTS_COUNT; i += offsetx) {
		generatePoint(&(Global->PointsArr[i]), rand_arr, i);
	}
}

__device__ void calculateLocalMin(GlobalData *Global, int32_t n, double t, double scale, double *max_arr) {
	int32_t i = getPixelX(Global->PointsArr[n].Pos.X, scale);
	int32_t j = getPixelY(Global->PointsArr[n].Pos.Y, scale);
	double x = getCoordinateX(i, scale);
	double y = getCoordinateY(j, scale);

	//if (func(i, j, t, scale))
	double funcRes = func(x, y, t);
	if (funcRes >= Global->PointsArr[n].LocalMin) {
		Global->PointsArr[n].LocalMin = funcRes;
		max_arr[n] = funcRes;
		Global->PointsArr[n].LocalMinPos.X = x;
		Global->PointsArr[n].LocalMinPos.Y = y;

	}
}

__device__ void changeParams(GlobalData *Global, int32_t n, double t, double scale) {
	int32_t i = getPixelX(Global->PointsArr[n].Pos.X, scale);
	int32_t j = getPixelY(Global->PointsArr[n].Pos.Y, scale);
	double x = getCoordinateX(i, scale);
	double y = getCoordinateY(j, scale);

	/*if (Global->PointsArr[n].LocalMin < Global->Min) {
		Global->Min = Global->PointsArr[n].LocalMin;
		Global->MinPos = Global->PointsArr[n].LocalMinPos;
	}*/

	Global->PointsArr[n].Speed = Global->PointsArr[n].Speed * INERTIA +
		PARAM_A_LOCAL * Global->PointSelectCoeff * distance(x, y,
			Global->PointsArr[n].LocalMinPos.X, Global->PointsArr[n].LocalMinPos.Y) +
		PARAM_A_GLOBAL * (1. - Global->PointSelectCoeff) * distance(x, y,
			Global->MinPos.X, Global->MinPos.Y);

	Position currLocalMin, currGlobalMin, resPos;
	setPosition(&currLocalMin, Global->PointsArr[n].LocalMinPos.X - x,
		Global->PointsArr[n].LocalMinPos.Y - y);
	setPosition(&currGlobalMin, Global->MinPos.X - x,
		Global->MinPos.Y - y);

	resPos.X = PARAM_A_LOCAL * Global->PointSelectCoeff * (currLocalMin.X - x) +
		PARAM_A_GLOBAL * (1. - Global->PointSelectCoeff) * (currGlobalMin.X - x);
	resPos.Y = PARAM_A_LOCAL * Global->PointSelectCoeff * (currLocalMin.Y - y) +
		PARAM_A_GLOBAL * (1. - Global->PointSelectCoeff) * (currGlobalMin.Y - y);

	double dist = distance(x, y, resPos.X, resPos.Y);
	resPos.X = resPos.X / dist * Global->PointsArr[n].Speed;
	resPos.Y = resPos.Y / dist * Global->PointsArr[n].Speed;

	Global->PointsArr[n].Pos.X += resPos.X;
	Global->PointsArr[n].Pos.Y += resPos.Y;
	
	//cuPrintf("Local: %lf ~ %lf : %lf\n", Global->PointsArr[n].LocalMin, Global->PointsArr[n].LocalMinPos.X, Global->PointsArr[n].LocalMinPos.Y);
}

__global__ void calculateLocalMinimums(GlobalData *Global, double t, double scale, double *max_arr) {
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t offsetx = blockDim.x * gridDim.x;

	for (int32_t n = idx; n < POINTS_COUNT; n += offsetx) {
		calculateLocalMin(Global, n, t, scale, max_arr);
	}
	__syncthreads();
	//cuPrintf("Global:\n");
}

__global__ void movePoints(GlobalData *Global, double t, double scale, double *max_arr, int32_t max_pos) {
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t offsetx = blockDim.x * gridDim.x;

	if (idx == 0) {
		Global->Min = max_arr[max_pos];
		Global->MinPos.X = Global->PointsArr[max_pos].Pos.X;
		Global->MinPos.Y = Global->PointsArr[max_pos].Pos.Y;
	}

	for (int32_t n = idx; n < POINTS_COUNT; n += offsetx) {
		changeParams(Global, n, t, scale);
	}
	//cuPrintf("Global:\n");
}

__host__ void generateRandValues() {
	hiprandGenerator_t rand_gen;
	hiprandCreateGenerator(&rand_gen ,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandDestroyGenerator(rand_gen);
	int32_t rand_arr[POINTS_COUNT * 3];
	int32_t *cuda_rand_arr;
	CSC(hipMalloc((void**) &cuda_rand_arr, sizeof(int32_t) * POINTS_COUNT * 3));
	//hiprandGenerateUniformDouble(rand_gen, rand_arr, POINTS_COUNT * 3);
	for (uint32_t i = 0; i < POINTS_COUNT * 3; i++) {
		rand_arr[i] = rand() % (int32_t)DOUBLE_GEN_ACCURACY;
	}
	CSC(hipMemcpy(cuda_rand_arr, rand_arr, sizeof(int32_t) * POINTS_COUNT * 3, hipMemcpyHostToDevice));

	setGlobalDataValues<<<blocks1D, threads1D>>>(GLOBAL, cuda_rand_arr);
	//CSC(hipFree(cuda_rand_arr));
}

struct hipGraphicsResource *res;
GLuint vbo;

double GLOBAL_SCALE = 1.;

__host__ int32_t findGlobalMaximum() {
	Comparator cmp;
	thrust::device_ptr <double> begin = thrust::device_pointer_cast(MAX_ARR);
	thrust::device_ptr <double> max = thrust::max_element(
		begin,
		begin + POINTS_COUNT, cmp);
	return max - begin;
}

void update() {
	static double t = 0.0;
	uchar4* dev_data;
	size_t size;
	CSC(hipGraphicsMapResources(1, &res, 0));
	cudaPrintfInit();
	CSC(hipGraphicsResourceGetMappedPointer((void**) &dev_data, &size, res));
	drawMap<<<blocks2D, threads2D>>>(GLOBAL, dev_data, t, GLOBAL_SCALE);
	drawPoints<<<blocks1D, threads1D>>>(GLOBAL, dev_data, t, GLOBAL_SCALE);
	cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
	CSC(hipGetLastError());
	CSC(hipGraphicsUnmapResources(1, &res, 0));
	glutPostRedisplay();
	t += 0.05;
	calculateLocalMinimums<<<blocks1D, threads1D>>>(GLOBAL, t, GLOBAL_SCALE, MAX_ARR);
	uint32_t max_pos = findGlobalMaximum();
	movePoints<<<blocks1D, threads1D>>>(GLOBAL, t, GLOBAL_SCALE, MAX_ARR, max_pos);
	GlobalData globalData;
	CSC(hipMemcpy(&globalData, GLOBAL, sizeof(GlobalData), hipMemcpyDeviceToHost));
	cout << globalData.Min << " ~ " << globalData.MinPos.X << " : " << globalData.MinPos.Y << endl; 
}

void display() {
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glDrawPixels(GLOBAL_WIDTH, GLOBAL_HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);	
	glutSwapBuffers();
}

void keys(unsigned char key, int32_t x, int32_t y) {
	if (key == 27) {
		CSC(hipGraphicsUnregisterResource(res));
		glBindBuffer(1, vbo);
		glDeleteBuffers(1, &vbo);
		destroyGlobalData();
		exit(0);
	}
	//cout << key << endl;
	if (key == '+') {
		GLOBAL_SCALE /= SCALE_CHANGE_SPEED;
		return;
	}
	if (key == '-') {
		GLOBAL_SCALE *= SCALE_CHANGE_SPEED;
		return;
	}
}


int main(int argc, char** argv) {
	
	setGlobalData();
	generateRandValues();


	glutInit(&argc, argv); 							
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);	
	glutInitWindowSize(GLOBAL_WIDTH, GLOBAL_HEIGHT);
	glutCreateWindow("Hot map");
	
	glutIdleFunc(update);
	glutDisplayFunc(display);
	glutKeyboardFunc(keys);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0.0, (GLdouble) GLOBAL_WIDTH, 0.0, (GLdouble) GLOBAL_HEIGHT);

	glewInit();

	glGenBuffers(1, &vbo);								
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, vbo);		
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, GLOBAL_WIDTH * GLOBAL_HEIGHT * sizeof(uchar4), NULL, GL_DYNAMIC_DRAW);

	CSC(hipGraphicsGLRegisterBuffer(&res, vbo, cudaGraphicsMapFlagsWriteDiscard));

	glutMainLoop();	
	return 0;
}