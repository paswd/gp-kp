#include "hip/hip_runtime.h"
// nvcc opengl.cu -lGL -lGLU -lGLEW -lglut

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>
#include <string.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include "../lib/cuPrintf.cu"

using namespace std;

const int32_t GLOBAL_WIDTH = 1024;
const int32_t GLOBAL_HEIGHT = 648;
const uint32_t VAR_COUNT = 2;
const uint32_t FUNC_MAXIMUMS_CNT = 3 ;
const uint32_t FUNC_MAXIMUMS_CNT_LIM = 5;
const uint32_t POINTS_COUNT = 500.;
const double SCALE_CHANGE_SPEED = 1.05;
const double PI = 3.1415926;
const double EPS = .00001;
const double DOUBLE_GEN_ACCURACY = 1000.;
const double POINTS_GEN_WIDTH = 10.;
const double POINTS_GEN_HEIGHT = 10.;
const double GRAVITY_PARAM = 10.;
const double DIST_LIM = 10.;
const double MOVING_PARAM_X = 5.;
const double MOVING_PARAM_Y = 5.;

const double INERTIA = .08;
const double PARAM_A_GLOBAL = .3;
const double PARAM_A_LOCAL = .2;

const double SHIFT_SPEED_X = .5;
const double SHIFT_SPEED_Y = .5;

/*const double FUNC_A[FUNC_MAXIMUMS_CNT][VAR_COUNT] = {
	{2.54, 6.35},
	{7.56, 3.35},
	{7.35, 3.65}
};*/

dim3 blocks2D(32, 32), threads2D(16, 16);
dim3 blocks1D(1024), threads1D(256);

#define CSC(call) {														\
	 hipError_t err = call;												\
	 if(err != hipSuccess) {											\
		  fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
				__FILE__, __LINE__, hipGetErrorString(err));			\
		  exit(1);														\
	 }																	\
} while (0)


const double 	xc = 0.0f,
				yc = 0.0f,
				sx = 5.0f,
				sy = sx * GLOBAL_HEIGHT / GLOBAL_WIDTH,
				FUNC_MIN = 0.,
				FUNC_MAX = .84;


struct Comparator {
	__host__ __device__ bool operator()(double a, double b) {
		return a < b;
	}
};

struct Position {
	double X;
	double Y;
};

__device__ __host__ void setPosition(Position *pos, double x, double y) {
	pos->X = x;
	pos->Y = y;
}

struct Point {
	Position Pos;

	double Angle;
	double Speed;
	double LocalMin;
	Position LocalMinPos;
	uchar4 Pixel;
	//bool PixelGradient;
};

struct GlobalData {
	Point *PointsArr;
	double Min;
	Position MinPos;
	double PointSelectCoeff;
	Position CurrCenter;
};

GlobalData *GLOBAL;
double *MAX_ARR;
double *POS_X;
double *POS_Y;

bool GLOBAL_MOVE = true;
bool MOVE_FUNCTION = false;

__host__ double fRand(double fMin, double fMax)
{
    double f = (double) (rand() % (int32_t) fMax);
    return fMin + f * (fMax - fMin);
}

__host__ void setGlobalData() {
	CSC(hipMalloc((void**) &GLOBAL, sizeof(GlobalData)));
	CSC(hipMalloc((void**) &MAX_ARR, sizeof(double) * POINTS_COUNT));
	CSC(hipMalloc((void**) &POS_X, sizeof(double) * POINTS_COUNT));
	CSC(hipMalloc((void**) &POS_Y, sizeof(double) * POINTS_COUNT));
	CSC(hipMemset(MAX_ARR, 0., sizeof(double) * POINTS_COUNT));

	Point *tmpPointsArr;
	CSC(hipMalloc((void**) &tmpPointsArr, sizeof(Point) * POINTS_COUNT));
	
	GlobalData globalData;
	globalData.PointsArr = tmpPointsArr;
	globalData.Min = 0.;
	globalData.CurrCenter.X = 0.;
	globalData.CurrCenter.Y = 0.;

	CSC(hipMemcpy(GLOBAL, &globalData, sizeof(GlobalData), hipMemcpyHostToDevice));
	//CURR_CENTER.X = 0.;
	//CURR_CENTER.Y = 0.;

}

__host__ void destroyGlobalData() {
	GlobalData globalData;
	CSC(hipMemcpy(&globalData, GLOBAL, sizeof(GlobalData), hipMemcpyDeviceToHost));
	Point *tmpPointsArr = globalData.PointsArr;

	CSC(hipFree(tmpPointsArr));
	CSC(hipFree(GLOBAL));
	CSC(hipFree(MAX_ARR));
	CSC(hipFree(POS_X));
	CSC(hipFree(POS_Y));
}

__device__ __host__ double distance(double x1, double y1, double x2, double y2) {
	return sqrt(pow(abs(x1 - x2), 2.) + pow(abs(y1 - y2), 2.));
}

__device__ __host__ int32_t distance(int32_t x1, int32_t y1, int32_t x2, int32_t y2) {
	return (abs(x1 - x2) + abs(y1 - y2));
}
	
__device__ __host__ double func(double x, double y, double t) {
	/*if (x < EPS && y < EPS) {
		x += EPS;
		y += EPS;
	}*/
	//return sin(x * x + t) + cos(y * y + t * 0.6) + sin(x * x + y * y + t * 0.3);
	//x /= 10.;
	//y /= 10.;
	double func_a[FUNC_MAXIMUMS_CNT_LIM][VAR_COUNT] = {
		//{0., 0.},
		{.054, 1.035},
		{3.956, .135},
		{.535, 1.065},
		{1.032, .121},
		{1.032, .121}
	};
	double summ = 0.;
	for (uint32_t i = 0; i < FUNC_MAXIMUMS_CNT; i++) {
		if (x - func_a[i][0] < EPS) {
			x += EPS;
		}
		if (y - func_a[i][1] < EPS) {
			y += EPS;
		}
		summ += 1. / (pow(x + MOVING_PARAM_X * cos(t * SHIFT_SPEED_X) - func_a[i][0], 2.) +
			pow(y + MOVING_PARAM_Y * sin(t * SHIFT_SPEED_Y) - func_a[i][1], 2.));
	}
	//summ = 2.5;
	//cout << summ << endl;
	//cuPrintf("%lf\n", summ);
	return summ;
}

/*__host__ double funcDbg(double x, double y, double t) {
	
	//return sin(x * x + t) + cos(y * y + t * 0.6) + sin(x * x + y * y + t * 0.3);
	//x /= 10.;
	//y /= 10.;
	double func_a[FUNC_MAXIMUMS_CNT_LIM][VAR_COUNT] = {
		//{0., 0.},
		{.054, 1.035},
		{3.956, .135},
		{.535, 1.065},
		{1.032, .121}
	};
	double summ = 0.;
	cout << "X = " << x << endl;
	cout << "XMod = " << (x - t * SHIFT_SPEED_X) << endl;
	cout << "XPrePow = " << (x - t * SHIFT_SPEED_X) - func_a[0][0] << endl;
	cout << "XPow = " << pow((x - t * SHIFT_SPEED_X) - func_a[0][0], 2.) << endl;
	cout << "Y = " << y << endl;
	cout << "YMod = " << (y - t * SHIFT_SPEED_Y) << endl;
	cout << "YPrePow = " << (y - t * SHIFT_SPEED_Y) - func_a[0][1] << endl;
	cout << "YPow = " << pow((y - t * SHIFT_SPEED_Y) - func_a[0][1], 2.) << endl;
	for (uint32_t i = 0; i < FUNC_MAXIMUMS_CNT; i++) {
		summ += 1. / (pow((x - t * SHIFT_SPEED_X) - func_a[i][0], 2.) + pow((y + t * SHIFT_SPEED_Y) - func_a[i][1], 2.));
	}
	//summ = 2.5;
	//cout << summ << endl;
	//cuPrintf("%lf\n", summ);
	return summ;
}*/

__device__ __host__ double getCoordinateX(int32_t i, double scale, Position shift) {
	return (2.0f * i / (double)(GLOBAL_WIDTH - 1) - 1.0f) * scale * sx + shift.X;
}

__device__ __host__ double getCoordinateY(int32_t j, double scale, Position shift) {
	return (2.0f * j / (double)(GLOBAL_HEIGHT - 1) - 1.0f) * scale * sy - shift.Y;
}

__device__ __host__ int32_t getPixelX(double x, double scale, Position shift) {
	return ((x - shift.X) / (2.0f * scale * sx) + 0.5f) * (double)(GLOBAL_WIDTH - 1);
}

__device__ __host__ int32_t getPixelY(double y, double scale, Position shift) {
	return ((y + shift.Y) / (2.0f * scale * sy) + 0.5f) * (double)(GLOBAL_HEIGHT - 1);
}

__device__ __host__ double func(int32_t i, int32_t j, double t, double scale, Position shift)  {
	return func(
		getCoordinateX(i, scale, shift),
		getCoordinateY(j, scale, shift),
		t);	 
}

__device__ __host__ bool isVisible(int32_t i, int32_t j) {
	return i > 0 && j > 0 && i < GLOBAL_WIDTH && j < GLOBAL_HEIGHT;
}

__device__ uchar4 get_color(float f) {
	//f /= 10;
	//f += .1;
	float k = 1.0 / 6.0;
	if (f < k)
		return make_uchar4((int)(f * 255 / k), 0, 0, 0);
	if (f < 2 * k)
		return make_uchar4(255, (int)((f - k) * 255 / k), 0, 0);
	if (f < 3 * k)
		return make_uchar4(255, 255, (int)((f - 2 * k) * 255 / k), 0);
	if (f < 4 * k)
		return make_uchar4(255 - (int)((f - 3 * k) * 255 / k), 255, 255, 0);
	if (f < 5 * k)
		return make_uchar4(0, 255 - (int)((f - 4 * k) * 255 / k), 255, 0);
	if (f < 6 * k)
		return make_uchar4(0, 0, 255 - (int)((f - 5 * k) * 255 / k), 0);
	return make_uchar4(0, 0, 0, 0);
}

__device__ uchar4 getPixel(double t) {
	return make_uchar4((int)(255 * cos(t + 2.)), (int)(255 * cos(t)), (int)(255 * sin(t)), 0);
}

/*__device__ uchar4 getPixel(GlobalData *Global, int32_t i, int32_t j, double f, double t, double scale) {
	for (int32_t n = 0; n < POINTS_COUNT; n++) {
		if (distance(getCoordinateX(i, scale), getCoordinateY(j, scale),
				Global->PointsArr[n].Pos.X, Global->PointsArr[n].Pos.Y) < POINT_RADIUS) {
			return getPixel(t);
		}
		if (distance(getCoordinateX(i, scale), getCoordinateY(j, scale),
				Global->PointsArr[n].Pos.X, Global->PointsArr[n].Pos.Y) < 2. * POINT_RADIUS) {
			return make_uchar4(0, 0, 0, 0);
		}
	}
	cuPrintf("UNIT %d :: %d\n", i, getPixelX(getCoordinateX(i, scale), scale));
	return get_color(f);
}*/


__global__ void drawMap(GlobalData *Global, uchar4* data, double t, double scale) {
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t idy = blockIdx.y * blockDim.y + threadIdx.y;
	int32_t offsetx = blockDim.x * gridDim.x;
	int32_t offsety = blockDim.y * gridDim.y;
	int32_t i, j;

	for (i = idx; i < GLOBAL_WIDTH; i += offsetx) {
		for (j = idy; j < GLOBAL_HEIGHT; j += offsety) {
			double f = (func(i, j, t, scale, Global->CurrCenter) - FUNC_MIN) / (FUNC_MAX - FUNC_MIN);
			//data[j * GLOBAL_WIDTH + i] = getPixel(Global, i, j, f, t, scale);
			data[j * GLOBAL_WIDTH + i] = get_color(f); //make_uchar4(0, 0, (int)(f * 255), 255);
		}
	}
	__syncthreads();
}

__global__ void drawPoints(GlobalData *Global, uchar4 *data, double t, double scale) {
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t offsetx = blockDim.x * gridDim.x;
	int32_t i, j;

	/*if (idx == 0) {
		int32_t i0 = getPixelX(0., scale);
		int32_t j0 = getPixelY(0., scale);
		int32_t i1 = getPixelX(1., scale);
		int32_t j1 = getPixelY(1., scale);

		data[j0 * GLOBAL_WIDTH + i0] = getPixel(t);
		data[j1 * GLOBAL_WIDTH + i1] = getPixel(t);
	}*/

	for (int32_t n = idx; n < POINTS_COUNT; n += offsetx) {
		i = getPixelX(Global->PointsArr[n].Pos.X, scale, Global->CurrCenter);
		j = getPixelY(Global->PointsArr[n].Pos.Y, scale, Global->CurrCenter);

		if (isVisible(i, j)) {
			data[j * GLOBAL_WIDTH + i] = getPixel(t);
		}
		//cuPrintf("Unit: %lf :: %lf :: %d\n", getCoordinateX(i, scale), Global->PointsArr[n].Pos.X, i);
	}

	__syncthreads();
}

__device__ void generatePoint(Point *point, int32_t *rand_arr, int32_t num) {
	//hiprandCreateGenerator(&rand_gen ,HIPRAND_RNG_PSEUDO_DEFAULT);

	//point->Pos.X = cuRand() % GLOBAL_WIDTH;
	//point->Pos.Y = cuRand() % GLOBAL_HEIGHT;
	/*hiprandGenerateUniformDouble(rand_gen, &(point->Pos.X), 1);
	hiprandGenerateUniformDouble(rand_gen, &(point->Pos.Y), 1);
	hiprandGenerateUniformDouble(rand_gen, &(point->Angle), 1);*/
	

	point->Pos.X = (double)(rand_arr[3 * num]) / DOUBLE_GEN_ACCURACY * POINTS_GEN_WIDTH;
	point->Pos.Y = (double)(rand_arr[3 * num + 1]) / DOUBLE_GEN_ACCURACY * POINTS_GEN_HEIGHT;
	point->Angle = (double)(rand_arr[3 * num + 2]) / DOUBLE_GEN_ACCURACY * 2. * PI;

	if (rand_arr[3 * num] % 2) {
		point->Pos.X = -point->Pos.X;
	}
	if (rand_arr[3 * num + 1] % 2) {
		point->Pos.Y = -point->Pos.Y;
	}
	point->Speed = 1.;
	point->LocalMin = 0.;
	point->LocalMinPos.X = 0.;
	point->LocalMinPos.Y = 0.;
	point->Pixel = make_uchar4(0, 255, 0, 0);


	//hiprandDestroyGenerator(rand_gen);
}

__global__ void setGlobalDataValues(GlobalData *Global, int32_t *rand_arr) {
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t offsetx = blockDim.x * gridDim.x;

	Global->PointSelectCoeff = rand_arr[0] / DOUBLE_GEN_ACCURACY;
	Global->MinPos.X = 0.;
	Global->MinPos.Y = 0.;

	for (int32_t i = idx; i < POINTS_COUNT; i += offsetx) {
		generatePoint(&(Global->PointsArr[i]), rand_arr, i);
	}
}

__device__ void calculateLocalMin(GlobalData *Global, int32_t n, double t, double scale, double *max_arr) {
	int32_t i = getPixelX(Global->PointsArr[n].Pos.X, scale, Global->CurrCenter);
	int32_t j = getPixelY(Global->PointsArr[n].Pos.Y, scale, Global->CurrCenter);
	double x = Global->PointsArr[n].Pos.X;
	double y = Global->PointsArr[n].Pos.Y;

	//if (func(i, j, t, scale))
	Global->PointsArr[n].LocalMin *= INERTIA;
	max_arr[n] *= INERTIA;
	double funcRes = func(i, j, t, scale, Global->CurrCenter);
	if (funcRes >= Global->PointsArr[n].LocalMin) {
		Global->PointsArr[n].LocalMin = funcRes;
		max_arr[n] = funcRes;
		Global->PointsArr[n].LocalMinPos.X = x;
		Global->PointsArr[n].LocalMinPos.Y = y;

	}
}

__device__ void getCollisionVector(GlobalData *Global, Position *res, int32_t n) {
	res->X = 0.;
	res->Y = 0.;
	//Position curr;
	for (int32_t i = 0; i < n; i++) {
		if (i == n) {
			continue;
		}
		double dist = distance(Global->PointsArr[i].Pos.X, Global->PointsArr[i].Pos.Y,
			Global->PointsArr[n].Pos.X, Global->PointsArr[n].Pos.Y);
		/*curr.X = -(Global->PointsArr[i].Pos.X - Global->PointsArr[n].Pos.X) / pow(dist, 4.);
		curr.Y = -(Global->PointsArr[i].Pos.Y - Global->PointsArr[n].Pos.Y) / pow(dist, 4.);

		curr.X /= pow(dist, 4.);
		curr.Y /= pow(dist, 4.);*/

		res->X += -(Global->PointsArr[i].Pos.X - Global->PointsArr[n].Pos.X) / pow(dist, 4.);
		res->Y += -(Global->PointsArr[i].Pos.Y - Global->PointsArr[n].Pos.Y) / pow(dist, 4.);
	}
	res->X /= (double)(POINTS_COUNT - 1) * GRAVITY_PARAM;
	res->Y /= (double)(POINTS_COUNT - 1) * GRAVITY_PARAM;

	__syncthreads();
}

__device__ void changeParams(GlobalData *Global, int32_t n, double t, double scale) {
	double x = Global->PointsArr[n].Pos.X;
	double y = Global->PointsArr[n].Pos.Y;

	/*if (Global->PointsArr[n].LocalMin < Global->Min) {
		Global->Min = Global->PointsArr[n].LocalMin;
		Global->MinPos = Global->PointsArr[n].LocalMinPos;
	}*/

	Global->PointsArr[n].Speed = Global->PointsArr[n].Speed * INERTIA +
		PARAM_A_LOCAL * Global->PointSelectCoeff * distance(x, y,
			Global->PointsArr[n].LocalMinPos.X, Global->PointsArr[n].LocalMinPos.Y) +
		PARAM_A_GLOBAL * (1. - Global->PointSelectCoeff) * distance(x, y,
			Global->MinPos.X, Global->MinPos.Y);
	/*Global->PointsArr[n].Speed = Global->PointsArr[n].Speed * INERTIA +
		PARAM_A_LOCAL * 0. * distance(x, y,
			Global->PointsArr[n].LocalMinPos.X, Global->PointsArr[n].LocalMinPos.Y) +
		PARAM_A_GLOBAL * (1. - 0.) * distance(x, y,
			Global->MinPos.X, Global->MinPos.Y);*/

	Position currLocalMin, currGlobalMin, resPos, collision;
	setPosition(&currLocalMin, Global->PointsArr[n].LocalMinPos.X - x,
		Global->PointsArr[n].LocalMinPos.Y - y);
	setPosition(&currGlobalMin, Global->MinPos.X - x,
		Global->MinPos.Y - y);

	resPos.X = PARAM_A_LOCAL * Global->PointSelectCoeff * currLocalMin.X +
		PARAM_A_GLOBAL * (1. - Global->PointSelectCoeff) * currGlobalMin.X;
	resPos.Y = PARAM_A_LOCAL * Global->PointSelectCoeff * currLocalMin.Y +
		PARAM_A_GLOBAL * (1. - Global->PointSelectCoeff) * currGlobalMin.Y;

	double dist = distance(x, y, resPos.X, resPos.Y);
	resPos.X = resPos.X / dist * Global->PointsArr[n].Speed;
	resPos.Y = resPos.Y / dist * Global->PointsArr[n].Speed;

	getCollisionVector(Global, &collision, n);

	Global->PointsArr[n].Pos.X += resPos.X + collision.X;
	Global->PointsArr[n].Pos.Y += resPos.Y + collision.Y;
	
	//cuPrintf("Local: %lf ~ %lf : %lf\n", Global->PointsArr[n].LocalMin, Global->PointsArr[n].LocalMinPos.X, Global->PointsArr[n].LocalMinPos.Y);
}

__global__ void calculateLocalMinimums(GlobalData *Global, double t, double scale, double *max_arr) {
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t offsetx = blockDim.x * gridDim.x;

	for (int32_t n = idx; n < POINTS_COUNT; n += offsetx) {
		calculateLocalMin(Global, n, t, scale, max_arr);
	}
	__syncthreads();
	//cuPrintf("Global:\n");
}

__global__ void movePoints(GlobalData *Global, double t, double scale, double *max_arr, int32_t max_pos,
		double *pos_x, double *pos_y) {
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t offsetx = blockDim.x * gridDim.x;

	if (idx == 0) {
		Global->Min *= INERTIA;
	}

	if (idx == 0 && max_arr[max_pos] > Global->Min) {
		Global->Min = max_arr[max_pos];
		Global->MinPos.X = Global->PointsArr[max_pos].Pos.X;
		Global->MinPos.Y = Global->PointsArr[max_pos].Pos.Y;
	}

	__syncthreads();

	for (int32_t n = idx; n < POINTS_COUNT; n += offsetx) {
		if (distance(
				Global->PointsArr[n].Pos.X, Global->PointsArr[n].Pos.Y,
				Global->CurrCenter.X, Global->CurrCenter.Y) < DIST_LIM) {
			pos_x[n] = Global->PointsArr[n].Pos.X;
			pos_y[n] = Global->PointsArr[n].Pos.Y;
		} else {
			pos_x[n] = 0.;
			pos_y[n] = 0.;
		}
		changeParams(Global, n, t, scale);
	}
	//cuPrintf("Global:\n");
}

__host__ void generateRandValues() {
	hiprandGenerator_t rand_gen;
	hiprandCreateGenerator(&rand_gen ,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandDestroyGenerator(rand_gen);
	int32_t rand_arr[POINTS_COUNT * 3];
	int32_t *cuda_rand_arr;
	CSC(hipMalloc((void**) &cuda_rand_arr, sizeof(int32_t) * POINTS_COUNT * 3));
	//hiprandGenerateUniformDouble(rand_gen, rand_arr, POINTS_COUNT * 3);
	for (uint32_t i = 0; i < POINTS_COUNT * 3; i++) {
		rand_arr[i] = rand() % (int32_t)DOUBLE_GEN_ACCURACY;
	}
	CSC(hipMemcpy(cuda_rand_arr, rand_arr, sizeof(int32_t) * POINTS_COUNT * 3, hipMemcpyHostToDevice));

	setGlobalDataValues<<<blocks1D, threads1D>>>(GLOBAL, cuda_rand_arr);
	//CSC(hipFree(cuda_rand_arr));
}

struct hipGraphicsResource *res;
GLuint vbo;

double GLOBAL_SCALE = 1.;

__host__ int32_t findGlobalMaximum() {
	Comparator cmp;
	thrust::device_ptr <double> begin = thrust::device_pointer_cast(MAX_ARR);
	thrust::device_ptr <double> max = thrust::max_element(
		begin,
		begin + POINTS_COUNT, cmp);
	return max - begin;
}

__global__ void calculateSumm(double *arr) {
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t offsetx = blockDim.x * gridDim.x;

	for (int32_t width = 2; width < POINTS_COUNT; width *= 2) {
		for (int32_t i = idx * width; i < POINTS_COUNT; i += offsetx * width) {
			if (i + width / 2 < POINTS_COUNT) {
				arr[i] += arr[i + width / 2];
			}
		}
		__syncthreads();
	}
}
__host__ void calculateSummTest(double *arr) {
	int32_t idx = 0;
	int32_t offsetx = 1;

	cout << "START" << endl;
	for (int32_t width = 2; width < POINTS_COUNT; width *= 2) {
		for (int32_t id = 0; id < POINTS_COUNT; id++) {
			cout << arr[id] << " ";
		}
		cout << endl;
		for (int32_t i = idx * width; i < POINTS_COUNT; i += offsetx * width) {
			if (i + width / 2 < POINTS_COUNT) {
				cout << i << " <- " << i + width / 2 << endl;
				cout << arr[i] << " + " << arr[i + width / 2];
				arr[i] += arr[i + width / 2];
				cout << " = " << arr[i] << endl;
				cout << "------------------" << endl;
			}
		}
		cout << "===================" << endl;
	}
	cout << "END" << endl << endl;
}

void updateCenter() {
	//thrust::device_ptr <double> begin_x = thrust::device_pointer_cast(POS_X);
	//thrust::device_ptr <double> begin_y = thrust::device_pointer_cast(POS_Y);
	//calculateSumm<<<blocks1D, threads1D>>>(POS_X);
	//calculateSumm<<<blocks1D, threads1D>>>(POS_Y);
	/*double pos_x[POINTS_COUNT];
	double pos_y[POINTS_COUNT];

	CSC(hipMemcpy(pos_x, POS_X, sizeof(double) * POINTS_COUNT, hipMemcpyDeviceToHost));
	CSC(hipMemcpy(pos_y, POS_Y, sizeof(double) * POINTS_COUNT, hipMemcpyDeviceToHost));*/

	/*calculateSummTest(pos_x);
	calculateSummTest(pos_y);*/
	cout << "TEST" << endl;
	thrust::device_vector<double> vect_x(POS_X, POS_X + POINTS_COUNT);
	thrust::device_vector<double> vect_y(POS_Y, POS_Y + POINTS_COUNT);
	double summ_x = thrust::reduce(vect_x.begin(), vect_x.end());
	double summ_y = thrust::reduce(vect_y.begin(), vect_y.end());
	//summ_x = 0.;
	//summ_y = 0.;
	//cout << summ_x << " " << summ_y << endl;
	//CSC(hipMemcpy(&summ_x, POS_X, sizeof(double), hipMemcpyDeviceToHost));
	//CSC(hipMemcpy(&summ_y, POS_Y, sizeof(double), hipMemcpyDeviceToHost));

	Position new_center;
	setPosition(
		&new_center,
		summ_x / POINTS_COUNT,
		summ_y / POINTS_COUNT
		);
	cout << new_center.X << " : " << new_center.Y << endl;
	GlobalData globalData;
	CSC(hipMemcpy(&globalData, GLOBAL, sizeof(GlobalData), hipMemcpyDeviceToHost));
	if (!isVisible(
		getPixelX(new_center.X, GLOBAL_SCALE, globalData.CurrCenter),
		getPixelY(new_center.Y, GLOBAL_SCALE, globalData.CurrCenter)
		)) {
		globalData.CurrCenter = new_center;
		CSC(hipMemcpy(GLOBAL, &globalData, sizeof(GlobalData), hipMemcpyHostToDevice));
		cout << "INVISIBLE" << endl;
	} else {
		cout << "  VISIBLE" << endl;
	}
}

void Test() {
	GlobalData globalData;
	CSC(hipMemcpy(&globalData, GLOBAL, sizeof(GlobalData), hipMemcpyDeviceToHost));
	Point *tmpPointsArr = globalData.PointsArr;
	Point pointArr[POINTS_COUNT];

	CSC(hipMemcpy(pointArr, tmpPointsArr, sizeof(Point) * POINTS_COUNT, hipMemcpyDeviceToHost));

	//double localMax[POINTS_COUNT];

	//CSC(hipMemcpy(localMax, MAX_ARR, sizeof(double) * POINTS_COUNT, hipMemcpyDeviceToHost));
	cout << "GLOBAL: " << globalData.Min << " ~ " << globalData.MinPos.X << " : " << globalData.MinPos.Y << endl;
	for (int32_t i = 0; i < POINTS_COUNT; i++) {
		cout << pointArr[i].Pos.X << " :: " << pointArr[i].Pos.Y << " - " << func(pointArr[i].Pos.X, pointArr[i].Pos.Y, 0)<< " ~~~ " << pointArr[i].LocalMinPos.X << "\t" << pointArr[i].LocalMinPos.Y << "\t" <<
			distance(pointArr[i].LocalMinPos.X, pointArr[i].LocalMinPos.Y, 0., 0.) << "\t" << pointArr[i].LocalMin << endl;
	}
}

void update() {
	static double t = 0.0;
	uchar4* dev_data;
	size_t size;
	CSC(hipGraphicsMapResources(1, &res, 0));
	cudaPrintfInit();
	CSC(hipGraphicsResourceGetMappedPointer((void**) &dev_data, &size, res));
	drawMap<<<blocks2D, threads2D>>>(GLOBAL, dev_data, t, GLOBAL_SCALE);
	drawPoints<<<blocks1D, threads1D>>>(GLOBAL, dev_data, t, GLOBAL_SCALE);
	cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
	CSC(hipGetLastError());
	CSC(hipGraphicsUnmapResources(1, &res, 0));
	glutPostRedisplay();
	if (GLOBAL_MOVE) {
		if (MOVE_FUNCTION) {
			t += 0.05;
		}
		calculateLocalMinimums<<<blocks1D, threads1D>>>(GLOBAL, t, GLOBAL_SCALE, MAX_ARR);
		//Test();
		uint32_t max_pos = findGlobalMaximum();
		movePoints<<<blocks1D, threads1D>>>(GLOBAL, t, GLOBAL_SCALE, MAX_ARR, max_pos, POS_X, POS_Y);
		updateCenter();
	}
	GlobalData globalData;
	CSC(hipMemcpy(&globalData, GLOBAL, sizeof(GlobalData), hipMemcpyDeviceToHost));
	//cout << globalData.Min << " ~ " << globalData.MinPos.X << " : " << globalData.MinPos.Y << endl; 
}

void display() {
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glDrawPixels(GLOBAL_WIDTH, GLOBAL_HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);	
	glutSwapBuffers();
}

void keys(unsigned char key, int32_t x, int32_t y) {
	if (key == 27) {
		CSC(hipGraphicsUnregisterResource(res));
		glBindBuffer(1, vbo);
		glDeleteBuffers(1, &vbo);
		destroyGlobalData();
		exit(0);
	}
	//cout << key << endl;
	if (key == '+') {
		GLOBAL_SCALE /= SCALE_CHANGE_SPEED;
		return;
	}
	if (key == '-') {
		GLOBAL_SCALE *= SCALE_CHANGE_SPEED;
		return;
	}
	if (key == 't') {
		Test();
		return;
	}
	if (key == 'p') {
		GLOBAL_MOVE = !GLOBAL_MOVE;
		return;
	}
	if (key == 'm') {
		MOVE_FUNCTION = !MOVE_FUNCTION;
		return;
	}
	//cout << (int) key << endl;
}


int main(int argc, char** argv) {
	//srand(time(NULL));
	//cout <<  "MAX = " << func(.054, 1.035, 0.) << " ~ " << .054 << " : " << 1.035 << endl;
	/*int n;
	cin >> n;*/
	setGlobalData();
	generateRandValues();


	glutInit(&argc, argv); 							
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);	
	glutInitWindowSize(GLOBAL_WIDTH, GLOBAL_HEIGHT);
	glutCreateWindow("Hot map");
	
	glutIdleFunc(update);
	glutDisplayFunc(display);
	glutKeyboardFunc(keys);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0.0, (GLdouble) GLOBAL_WIDTH, 0.0, (GLdouble) GLOBAL_HEIGHT);

	glewInit();

	glGenBuffers(1, &vbo);								
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, vbo);		
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, GLOBAL_WIDTH * GLOBAL_HEIGHT * sizeof(uchar4), NULL, GL_DYNAMIC_DRAW);

	CSC(hipGraphicsGLRegisterBuffer(&res, vbo, cudaGraphicsMapFlagsWriteDiscard));

	glutMainLoop();	
	return 0;
}